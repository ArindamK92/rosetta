#include "hip/hip_runtime.h"
//#include "pointwise.h"
#include "rosetta.h"
//#include <benchmark/benchmark.h>
//#include <cstdlib>
//#include <string>
//#include "synchronization.hpp" 

// Loosely based on CUDA Toolkit sample: vectorAdd

__global__ void kernel(int n, double *A) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
        A[i] += 42;
}


 void run(State& state, int n) {
    // default size
    if (n < 0)
        n = (DEFAULT_N);

    double *A = state.malloc<double>(n);   


   // cuptiSubscribe() ;

//hipEvent_t start, stop;
//hipEventCreate(&start);
//hipEventCreate(&stop);

    double *dev_A;
    BENCH_CUDA_TRY(hipMalloc((void**)&dev_A, n * sizeof(double)));


    hipMemcpy(dev_A, A, n * sizeof(double), hipMemcpyHostToDevice);



// TODO: dim3 dimBlock(16, 16, 1);
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

//state.PauseTiming();
    for (auto &&_ : state.manual()) {
//state.PauseTiming();
       hipStream_t stream = 0;
    //    cuda_event_timer raii(state, true, stream); 
//state.ResumeTiming();

{
        auto &&scope = _.scope();
        kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(n, dev_A);
        }

        // TODO: Is the invocation already blocking?
        hipMemcpy( A, dev_A, n * sizeof(double), hipMemcpyDeviceToHost ); hipDeviceSynchronize();

    }

    hipFree(dev_A);


    state.verifydata(A, n);
    state.free(A);
}


#if 0
int main(int argc, char* argv[]) {
    ::benchmark::Initialize(&argc, argv);

    int n = N;
    if (argc > 1) {
       n = std::atoi(argv[1]);
       argc -= 1;
       argv += 1;
    }

    benchmark::RegisterBenchmark(("pointwise.cuda" + std::string("/") +std:: to_string(n) + "/gpu").c_str(), &pointwise_cuda, n)->MeasureProcessCPUTime()->UseRealTime()->Unit(benchmark::kMillisecond)->UseManualTime();
    benchmark::RegisterBenchmark(("pointwise.cuda" + std::string("/") +std:: to_string(n) + "/cpu").c_str() , &pointwise_cuda, n)->MeasureProcessCPUTime()->UseRealTime()->Unit(benchmark::kMillisecond);

    if (::benchmark::ReportUnrecognizedArguments(argc, argv)) return 1;
    ::benchmark::RunSpecifiedBenchmarks();
    ::benchmark::Shutdown();
    return EXIT_SUCCESS;
}
#endif

