#include "hip/hip_runtime.h"
// BUILD: add_benchmark(ppm=cuda)

#include <rosetta.h>



__global__ void cuda_assign(pbsize_t n, idx_t i, real *data) {
  idx_t i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < n) 
    data[i] = std::sqrt(data[i]);
}

static int num_blocks(int num, int factor) {
  return (num + factor - 1) / factor;
}





void run(State &state, pbsize_t pbsize) {
  pbsize_t n = pbsize; 


  auto data = state.allocate_array<real>({m, n}, /*fakedata*/ false, /*verify*/ true, "data");



  real *dev_data = state.allocate_dev<real>(n);




  for (auto &&_ : state) {
    CUDA_CHECK(   hipMemcpy(dev_data, data.data(), n * sizeof(real), hipMemcpyHostToDevice) );

    const int threadsPerBlock = 256;
    cuda_assign<<<threadsPerBlock, num_blocks(m, threadsPerBlock)>>>(n, dev_data);
  
    CUDA_CHECK( hipMemcpy(data.data(), dev_data, n * sizeof(real), hipMemcpyDeviceToHost);
    CUDA_CHECK(hipDeviceSynchronize());
  }


  state.free_dev(dev_data);
}
