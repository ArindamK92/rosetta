#include "hip/hip_runtime.h"
// BUILD: add_benchmark(ppm=cuda)

#include <rosetta.h>


__global__ void cuda_assign(pbsize_t n, real *data) {
  idx_t i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < n)
    data[i] = i;
}


static int num_blocks(int num, int factor) {
  return (num + factor - 1) / factor;
}


void run(State &state, pbsize_t pbsize) {
  pbsize_t n = pbsize;


  auto data = state.allocate_array<real>({n}, /*fakedata*/ false, /*verify*/ true, "data");
  real *dev_data = state.allocate_dev<real>(n);

  for (auto &&_ : state) {
    const int threadsPerBlock = 256;
    cuda_assign<<<num_blocks(n, threadsPerBlock), threadsPerBlock>>>(n, dev_data);

    BENCH_CUDA_TRY(hipMemcpy(data.data(), dev_data, n * sizeof(real), hipMemcpyDeviceToHost));
    BENCH_CUDA_TRY(hipDeviceSynchronize());
  }

  state.free_dev(dev_data);
}
