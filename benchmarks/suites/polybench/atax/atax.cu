#include "hip/hip_runtime.h"
// BUILD: add_benchmark(ppm=cuda)
#include "rosetta.h"







__global__ void kernel3(int m, int n, real * A, real *x, real *y, real *tmp) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if ( i >= m || j >= n) return; 
            tmp[i] += A[i * m + j] * x[j];
}


__global__ void kernel4(int m, int n, real * A, real *x, real *y, real *tmp) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if ( i >= m || j >= n) return; 

            y[j] +=   A[i*m +j] * tmp[i];
}


int num_blocks(int num, int factor) {
    return (num + factor -1)/factor ;
}


void run(State& state, int pbsize) {
    // n is 5%-20% larger than m
    pbsize_t n = pbsize;
    pbsize_t m = pbsize - pbsize / 10;

    auto A = state.allocate_array<real>({n, m}, /*fakedata*/ true, /*verify*/ false);
    auto x = state.allocate_array<real>({n}, /*fakedata*/ false, /*verify*/ false);
    auto y = state.allocate_array<real>({n}, /*fakedata*/ false, /*verify*/ true);
    auto tmp = state.allocate_array<real>({m}, /*fakedata*/ false, /*verify*/ false);


    real *dev_A, *dev_x, *dev_y, *dev_tmp;
    BENCH_CUDA_TRY(hipMalloc((void**)&dev_A, n * m * sizeof(real))); // TODO: Runtime should do this
    BENCH_CUDA_TRY(hipMalloc((void**)&dev_x, n *  sizeof(real)));
    BENCH_CUDA_TRY(hipMalloc((void**)&dev_y, n *  sizeof(real)));
    BENCH_CUDA_TRY(hipMalloc((void**)&dev_tmp, n *  sizeof(real)));

   


    int threadsPerBlock = 256;
    dim3 block (threadsPerBlock/32, 32, 1);
    dim3 grid (num_blocks(m,block.x), num_blocks(n,block.y), 1); 
   


    for (auto &&_ : state.manual()) {
        

        {
            auto &&scope = _.scope();

            hipMemcpy(dev_A, A.data(), n * m * sizeof(real), hipMemcpyHostToDevice);
            hipMemset(dev_y, 0, n * sizeof(real) );
            hipMemset(dev_tmp, 0, m * sizeof(real));

            kernel3<<<grid, block>>>(m,n,dev_A,dev_x, dev_y, dev_tmp);
            kernel4<<<grid, block>>>(m,n,dev_A,dev_x, dev_y, dev_tmp);

            hipMemcpy( dev_y, y.data() , n * sizeof(double), hipMemcpyDeviceToHost ); 
        }

        // TODO: Is the invocation already blocking?
       hipDeviceSynchronize();

    }

    BENCH_CUDA_TRY(   hipFree(dev_A));
    BENCH_CUDA_TRY(   hipFree(dev_x));
    BENCH_CUDA_TRY(   hipFree(dev_y));
    BENCH_CUDA_TRY(   hipFree(dev_tmp));
}


