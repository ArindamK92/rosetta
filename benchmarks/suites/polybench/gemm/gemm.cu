#include "hip/hip_runtime.h"
// BUILD: add_benchmark(ppm=cuda)

#include "rosetta.h"



__global__ void kernel_dev(pbsize_t ni, pbsize_t nj, pbsize_t nk,
                           real alpha,
                           real beta,
                           real *C, real *A, real *B) {
  idx_t i = blockDim.x * blockIdx.x + threadIdx.x;
  idx_t j = blockDim.y * blockIdx.y + threadIdx.y;


  if (i < ni && j < nj) {
    C[i * nj + j] *= beta;


    for (idx_t k = 0; k < nk; k++)
      C[i * nj + j] += alpha * A[i * nk + k] * B[k * nj + j];
  }
}



static unsigned num_blocks(int num, int factor) {
  return (num + factor - 1) / factor;
}



static void kernel(pbsize_t ni, pbsize_t nj, pbsize_t nk,
                   real alpha,
                   real beta,
                   real *C, real *A, real *B) {

  unsigned threadsPerBlock = 256;
  dim3 block{threadsPerBlock / 32, 32, 1};
  dim3 grid{num_blocks(ni, block.x), num_blocks(nj, block.y), 1};
  kernel_dev<<<block, grid>>>(ni, nj, nk, alpha, beta, C, A, B);
}


void run(State &state, pbsize_t pbsize) {
  pbsize_t ni = pbsize - pbsize / 4;
  pbsize_t nj = pbsize - pbsize / 8;
  pbsize_t nk = pbsize;

  real alpha = 1.5;
  real beta = 1.2;
  auto C = state.allocate_array<real>({ni, nj}, /*fakedata*/ true, /*verify*/ true, "C");
  auto A = state.allocate_array<real>({ni, nk}, /*fakedata*/ true, /*verify*/ false, "A");
  auto B = state.allocate_array<real>({nk, nj}, /*fakedata*/ true, /*verify*/ false, "B");


  real *dev_C = state.allocate_dev<real>(ni * nj);
  real *dev_A = state.allocate_dev<real>(ni * nk);
  real *dev_B = state.allocate_dev<real>(nk * nj);

  for (auto &&_ : state) {
    hipMemcpy(dev_C, C.data(), ni * nj * sizeof(real), hipMemcpyHostToDevice);
    hipMemcpy(dev_A, A.data(), ni * nk * sizeof(real), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B.data(), nk * nj * sizeof(real), hipMemcpyHostToDevice);



    kernel(ni, nj, nk, alpha, beta, dev_C, dev_A, dev_B);


    hipMemcpy(C.data(), dev_C, ni * nj * sizeof(real), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
  }

  state.free_dev(dev_C);
  state.free_dev(dev_A);
  state.free_dev(dev_B);
}
